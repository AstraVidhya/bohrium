
#include <hip/hip_runtime.h>
/*
 * Copyright 2011 Troels Blum <troels@blum.dk>
 *
 * This file is part of cphVB <http://code.google.com/p/cphvb/>.
 *
 * cphVB is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * cphVB is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with cphVB. If not, see <http://www.gnu.org/licenses/>.
 */

extern "C" __global__ 
void add_reduce_float_1d(float* in, uint elements, float* out)
{
    const uint nThreads = blockDim.x*gridDim.x;
	const uint outIdx = threadIdx.x + blockIdx.x*blockDim.x;
    float myRes = 0.0;
    uint inIdx;
	for (inIdx = outIdx; inIdx < elements; inIdx += nThreads) 
	{
        myRes += in[inIdx];
	}
    out[outIdx] = myRes;
}


